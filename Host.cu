#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <unordered_map>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "./data_utilities/StatStructs.h"
#include <limits>
#include <future>
#include "./lda/LDA.cpp"
#include <iomanip>
#include <algorithm>
#include <string>
#include <vector>
#include <map>
#include <cmath>
#include <tuple>
#include "./hyperblock_generation/MergerHyperBlock.cuh"
#include "./hyperblock/HyperBlock.h"
#include "./interval_hyperblock/IntervalHyperBlock.h"
#include "./knn/Knn.h"
#include "./screen_output/PrintingUtil.h"
#include "./data_utilities/DataUtil.h"
#include "./simplifications/Simplifications.h"
#include "classification_testing/ClassificationTests.h"
using namespace std;

#ifdef _WIN32
    const string PATH_SEPARATOR = "\\";
#else
    const string PATH_SEPARATOR = "/";
#endif

#define LDA_ORDERING true

int NUM_CLASSES;   // Number of classes in the dataset
int NUM_POINTS;    // Total number of points in the dataset
int FIELD_LENGTH;  // Number of attributes in the dataset
int COMMAND_LINE_ARGS_CLASS = -1; // used for when we are splitting up generation one class per machine. This lets us run on many computers at once.

// USED SO THAT WE CAN GET THE NAMES OF EACH CLASS BASICALLY FOR TRAIN AND TEST DATA
map<string, int> CLASS_MAP;
map<string, int> CLASS_MAP_TESTING;
map<int, string> CLASS_MAP_INT;
map<int, string> CLASS_MAP_TESTING_INT;

void evaluateOneToOneHyperBlocks(const vector<vector<HyperBlock>>& oneToOneHBs,const vector<vector<vector<float>>>& testSet,const vector<pair<int, int>>& classPairs, int numClasses);

/**
 * For each class of data that we have:
 * Generate "class" and "not class" HBs
 *
 * We will train each class on the opposing points from ALL other classes.
 * For example: Generate HBS for class 0 using the counter points from classes 1,2,3
 *              then generate HBS for (1,2,3) together against counter points from class 0.
 *
 */
vector<vector<HyperBlock>> oneToRestHyper(const vector<vector<vector<float>>>& trainSet, vector<int> ecBestVecIdx) {
    vector<vector<HyperBlock>> oneToRestBlocks;
    vector<HyperBlock> tempHBs;

    const int numClasses = trainSet.size();

    for(int i = 0; i < numClasses; i++) {
        cout << "Training Class (REAL LABEL): " << CLASS_MAP_INT[i] << endl;
        tempHBs.clear();

        // Train with this class as first class, and ALL others as the second class.
        vector<vector<vector<float>>> trainingData(2);
        trainingData[0] = trainSet[i];

        // Add all other class data as the "second class"
        for(int j = 0; j < numClasses; j++) {
            if(j == i) continue;
            for(const auto& point : trainSet[j]) trainingData[1].push_back(point);
        }

        vector<int> bestVecs(2);
        bestVecs[0] = ecBestVecIdx[i];
        bestVecs[1] = ecBestVecIdx[i];

        // Now we generate "HBs for class i" and "HBs for not-class i"
        IntervalHyperBlock::generateHBs(trainingData, tempHBs, bestVecs, FIELD_LENGTH, -1);
        oneToRestBlocks.push_back(tempHBs);

        // Set the HBs generated to have the correct class number.
        for(auto& hb : oneToRestBlocks[i]) {
            if(hb.classNum == 0) {
                hb.classNum = i;                    // Set it to be the correct class index i
                continue;
            }

            hb.classNum = numClasses + i;   // Doing this to be safe in case doing -i for the class would cause indexing issues anywhere.
        }
    }


    return oneToRestBlocks;
}


/* If we have classes a, b, c. then we should generate blocks to change this to a multi-step 2 class problem
   we will make blocks for a pair at a time.

   a vs b, a vs c, b vs c.

   We will now be able to pass a test set point to be evaluated by each set INDEPENDENTLY.
   We will either treat it as a sieve, ex use the set of blocks that is LEAST over-generalized.
   Future Work: or it can be treated as a voting type of system ex point p gets evaluated on ALL sets
   if it scores a, c, c for the 3 pairs specified above we might want to put it in class c.

   https://machinelearningmastery.com/one-vs-rest-and-one-vs-one-for-multi-class-classification/
*/
vector<vector<HyperBlock>> oneToOneHyper(const vector<vector<vector<float>>>& trainingData, vector<int> eachClassBestVectorIndex, vector<pair<int,int>> &classPairs){
      vector<vector<HyperBlock>> oneToOneHyperBlocks;
      vector<HyperBlock> pairHyperBlocks;
      const int numClasses = trainingData.size();
      cout << "Num Classes" << endl;

    // Make the call to interval hyper FOR EACH set
      for(int i = 0; i < numClasses; i++){
          for(int j = i + 1; j < numClasses; j++){
              pairHyperBlocks.clear();
              cout << i << ", " << j << endl;
              vector<vector<vector<float>>> binaryTrainingData(2);
              binaryTrainingData[0] = trainingData[i];
              binaryTrainingData[1] = trainingData[j];

              // Pass binary training data in . ex class 0 and 1.
              IntervalHyperBlock::generateHBs(binaryTrainingData, pairHyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);

              // Copy all the newly added blocks to the right index in the fin
              oneToOneHyperBlocks.push_back(pairHyperBlocks);
              classPairs.emplace_back(i, j);
              cout << "pDone" << classPairs.size() << endl;
          }
      }

      cout << "Done generating on-to-one HBs, now we are setting their classes to be correct..." << endl;
      // Now what we do.
      for(int i = 0; i < oneToOneHyperBlocks.size(); i++){
         for(int j = 0; j < oneToOneHyperBlocks[i].size(); j++){
            HyperBlock& hb = oneToOneHyperBlocks[i].at(j);

            hb.classNum = (hb.classNum == 0) ? classPairs[i].first : classPairs[i].second;
         }
      }

    return oneToOneHyperBlocks;
}

float testAccuracyOfHyperBlocks(vector<HyperBlock> &hyperBlocks, vector<vector<vector<float>>> &testData, vector<vector<vector<float>>> &trainingData, map<pair<int, int>, PointSummary>& pointSummaries, int k = 5, float threshold = 0.25) {

    // get our confusion matrix by just classifying with the blocks like normal
    vector<vector<vector<float>>> notClassifiedPoints(NUM_CLASSES);
    vector<vector<long>> hyperBlocksConfusionMatrix = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, testData, ClassificationTests::HYPERBLOCKS, notClassifiedPoints, NUM_CLASSES, pointSummaries);

    cout << "------------------------HYPERBLOCKS CONFUSION MATRIX-----------------------------" << endl;
    float hbAccuracy = PrintingUtil::printConfusionMatrix(hyperBlocksConfusionMatrix, NUM_CLASSES, CLASS_MAP_INT);

    // now build our second confusion matrix out of the unclassified stuff only
    vector<vector<vector<float>>> stillNotClassifiedPoints(NUM_CLASSES);
    vector<vector<long>> knnMatrix = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, notClassifiedPoints, ClassificationTests::PURE_KNN, stillNotClassifiedPoints, NUM_CLASSES, pointSummaries, k, threshold);

    cout << "------------------------KNN CONFUSION MATRIX--------------------------------" << endl;
    float knnAccuracy = PrintingUtil::printConfusionMatrix(knnMatrix, NUM_CLASSES, CLASS_MAP_INT);

    vector<vector<long>> finalConfusionMatrix(NUM_CLASSES, vector<long>(NUM_CLASSES, 0));
    // now we can just combine the two matrices
    for (int i = 0; i < NUM_CLASSES; i++) {
        for (int j = 0; j < NUM_CLASSES; j++) {
            finalConfusionMatrix[i][j] = knnMatrix[i][j] + hyperBlocksConfusionMatrix[i][j];
        }
    }

    cout << "-------------------------FINAL (HB + KNN predictions) CONFUSION MATRIX--------------------------" << endl;
    float finalAccuracy = PrintingUtil::printConfusionMatrix(finalConfusionMatrix, NUM_CLASSES, CLASS_MAP_INT);
    return finalAccuracy;
}

/* This function computes the LDA ordering for a given training dataset.
 * It sets up the bestVectors, bestVectorsIndexes, and eachClassBestVectorIndex.
 * best vectors is the weights of each coefficient from the LDF function
 * bestVectorsIndexes is just the indexes that correspond to those weights from the function, since we are sorting them
 * eachClassBestVectorIndex is the one best attribute for each class, we sort by this when generating blocks, and it helps a bit.
*
 * returns the class accuracy ordering so we can use the "sift approach" for HBs ex {0, 4, 2, 1, 5, 6, 7, 8, 9} for MNIST classes 0 is best seperable, 9 would be worst.
 */
vector<int> computeLDAOrdering(const vector<vector<vector<float>>>& trainingData, vector<vector<float>>& bestVectors, vector<vector<int>>& bestVectorsIndexes, vector<int>& eachClassBestVectorIndex) {
    // Run LDA on the training data.
    pair<vector<vector<float>>, vector<int>> result = linearDiscriminantAnalysis(trainingData);
    vector<int> classOrder;
    bestVectors = result.first;

    // Accuracy ordering of the LDA on the training data.
    classOrder = result.second;

    // Resize our index containers.
    bestVectorsIndexes.assign(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
    eachClassBestVectorIndex.assign(NUM_CLASSES, 0);

    // For each class, initialize the indexes and then sort (if desired)
    // and determine the index with the largest absolute LDA coefficient.
    for (int i = 0; i < NUM_CLASSES; i++) {
        // Populate with initial indices: 0, 1, 2, ... FIELD_LENGTH - 1.
        for (int j = 0; j < FIELD_LENGTH; j++) {
            bestVectorsIndexes[i][j] = j;
        }

#ifdef LDA_ORDERING
        // Optionally sort the indexes for class i based on the absolute value of the LDA coefficients.
        sort(bestVectorsIndexes[i].begin(), bestVectorsIndexes[i].end(),
             [&](int a, int b) {
                 return fabs(bestVectors[i][a]) < fabs(bestVectors[i][b]);
             });
#endif
        // Find the index (from bestVectorsIndexes) corresponding to the largest absolute LDA coefficient.
        // We use the values in bestVectors[i] for comparison.
        auto it = max_element(bestVectorsIndexes[i].begin(), bestVectorsIndexes[i].end(),
                              [&](int a, int b) {
                                  return fabs(bestVectors[i][a]) < fabs(bestVectors[i][b]);
                              });
        eachClassBestVectorIndex[i] = distance(bestVectorsIndexes[i].begin(), it);
    }

    return classOrder;
}

/******************************************************************
 * 10‑fold sweep over (k, threshold) pairs
 * ‑ kVals   : list of neighbour counts to test
 * ‑ tVals   : list of similarity‑threshold multipliers to test
 * Returns {bestK, bestT, bestAcc}
 ******************************************************************/
tuple<int,float,float> findBestParameters(vector<vector<vector<float>>> &dataset, vector<int> kVals = vector<int>{3, 5, 7, 9}, vector<float> tVals = vector<float>{0.15f, 0.20f, 0.25f, 0.30f}, int removalCount = 5, bool hidePrinting = false, int blockLevel = 1) {

    if (dataset.empty()) {
        cerr<<"Empty dataset\n";
        return make_tuple(-1,-1.f,-1.f);
    }

    /* -------- silence console if requested -------- */
    streambuf *oldBuf=nullptr; ostringstream sink;
    if (hidePrinting) oldBuf = cout.rdbuf(sink.rdbuf());

    const int FOLDS = 10;
    auto folds = DataUtil::splitDataset(dataset,FOLDS);

    /* accuracy accumulator indexed by [kIdx][tIdx] */
    vector<vector<float>> acc(kVals.size(),vector<float>(tVals.size(),0.f));

    for (int i=0;i<FOLDS;++i)
    {
        /* -------- build train / test split -------- */
        vector<vector<vector<float>>> train(NUM_CLASSES), test  = folds[i];

        for (int f=0;f<FOLDS;++f)
            if (f!=i)
                for (int c=0;c<NUM_CLASSES;++c)
                    train[c].insert(train[c].end(),folds[f][c].begin(),folds[f][c].end());

        /* -------- generate & simplify blocks -------- */
        vector<HyperBlock> hbs;
        vector<vector<float>> bestVecs;
        vector<vector<int>> bestIdx(NUM_CLASSES,vector<int>(FIELD_LENGTH));
        vector<int> eachBest(NUM_CLASSES);
        computeLDAOrdering(train,bestVecs,bestIdx,eachBest);

        IntervalHyperBlock::generateHBs(train,hbs,eachBest, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
        Simplifications::REMOVAL_COUNT = removalCount;

        // make a copy of our input data so that we don't break it for the KNN.
        vector<vector<vector<float>>> levelNTrain = train;

        // increase our block level until we hit the level we want.
        for (int level = 1; level < blockLevel; level++) {
            vector<HyperBlock> newBlocks;

            levelNTrain = move(IntervalHyperBlock::generateNextLevelHBs(levelNTrain, hbs, newBlocks, eachBest, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS));

            // knn actually does better when we are removing the extra points sometimes, so we use the original data and shrink the set does better KNN
            // train = move(IntervalHyperBlock::generateNextLevelHBs(train, hbs, newBlocks, eachBest, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS));

            hbs = move(newBlocks);
        }

        Simplifications::runSimplifications(hbs,train,bestIdx);

        /* -------- evaluate every (k,threshold) combo -------- */
        for (size_t kI=0;kI<kVals.size();++kI)
            for (size_t tI=0;tI<tVals.size();++tI) {
                Knn::deviationsComputed = false;            // reset per fold
                map<pair<int,int>,PointSummary> summaries;
                float foldAcc = testAccuracyOfHyperBlocks(hbs, test, train, summaries, kVals[kI], tVals[tI]);
                acc[kI][tI] += foldAcc;
            }
    }

    /* -------- compute averages & find best -------- */
    int    bestK  = -1;
    float  bestT  = -1.f;
    float  bestAcc= -1.f;

    for (size_t kI=0;kI<kVals.size();++kI)
        for (size_t tI=0;tI<tVals.size();++tI)
        {
            float avg = acc[kI][tI] / static_cast<float>(FOLDS);
            if (avg > bestAcc)
            { bestAcc = avg; bestK = kVals[kI]; bestT = tVals[tI]; }
            cout<<"K="<<kVals[kI]<<"  T="<<tVals[tI]
                     <<"  avgAcc="<<avg<<"\n";
        }

    if (hidePrinting) cout.rdbuf(oldBuf);

    cout<<"BEST -> K = "<<bestK<<"\tThreshold = "<< bestT <<"\taccuracy = " << bestAcc <<"\n";
    return make_tuple(bestK,bestT,bestAcc);
}

vector<float> runKFold(vector<vector<vector<float>>> &dataset, vector<pair<int,int>>& classPairs, bool oneToMany = true, bool takeUserInput = false, int removalCount = 5, int nearestNeighborK = 5, float similarityThreshold = 0.25f, bool hidePrinting = false) {

    if (dataset.empty()) {
        cout << "Please enter a training dataset before using K Fold validation" << endl;
        return {-1, -1, -1};
    }

    int k;
    // if we're taking input run it like normal. using this variable lets us just do it this way.
    if (takeUserInput) {
        cout << "Please Enter a K value:\t";
        cin >> k;

        // Clear the newline from the input buffer.
        cin.ignore(numeric_limits<streamsize>::max(), '\n');

        if (cin.fail() || k < 2) {
            cout << "Error: Invalid input. Please enter a valid integer greater than 1." << endl;
            // Clear the error state and ignore any remaining input.
            cin.clear();
            cin.ignore(numeric_limits<streamsize>::max(), '\n');
            return {-1, -1, -1};
        }
    }
    // if we're not using user input, we are testing for best accuracy and we can use 10.
    else
        k = 10;


    // used to hide the printing of the regular kFold testing stuff. so that when we are finding best parameters we don't have all that printing
    streambuf* oldBuf = nullptr;
    if (hidePrinting) {
        ostringstream nullSink;
        oldBuf = cout.rdbuf(nullSink.rdbuf());   // silence everything
    }

    vector<vector<vector<vector<float>>>> kFolds = DataUtil::splitDataset(dataset, k);
    // stats trackers for cross folds.
    float acc = 0.0f;
    int blockCount = 0;
    int cCount = 0;

    // generate blocks with a training set which is all folds except i. using i as the test dataset.
    for (int i = 0; i < k; i++) {

        // trainingData will store all folds except the i-th as training data.
        vector<vector<vector<float>>> trainingData(NUM_CLASSES);

        // Loop through all folds except i and accumulate points by class.
        for (int fold = 0; fold < k; fold++) {
            if (fold == i) continue; // skip test fold

            // build our training data
            for (int cls = 0; cls < NUM_CLASSES; cls++) {
                // Append all points from kFolds[fold][cls] to trainingData[cls]
                trainingData[cls].insert(trainingData[cls].end(), kFolds[fold][cls].begin(), kFolds[fold][cls].end());
            }
        }

        // little thing. causes issues if we don't reset when we make a new training set
        Knn::deviationsComputed = false;

        // The test dataset for this iteration is simply fold i.
        vector<vector<vector<float>>> testData = kFolds[i];

        // now that our data is set up with training and testing, we simply do business as usual. we are going to do our LDA on the train data, then just do our block generation and simplification
        // Run LDA on the training data.
        vector<vector<float>>bestVectors;

        // Initialize indexes for each class
        vector<vector<int>> bestVectorsIndexes = vector<vector<int> >(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
        vector<int> eachClassBestVectorIndex = vector<int>(NUM_CLASSES);

        computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);

        cout << "----------------------------FOLD " << (i + 1) << " RESULTS----------------------------------" << endl;
        vector<HyperBlock> hyperBlocks;

        if (oneToMany) {
            // ------------------------------------------
            // GENERATING BLOCKS BUSINESS AS USUAL
            IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);

            // simplify them, with the simplification count we have specifed as a parameter. usually 0, but playing with this value can get us better results because we are removing more blocks
            Simplifications::REMOVAL_COUNT = removalCount;
            //vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);

            // clause count computed here because sometimes we don't simplify
            int totalPoints = 0;
            for (const auto &c : trainingData)
                totalPoints += c.size();

            int clauseCount = 0;
            for (const auto &hb : hyperBlocks) {
                for (int a = 0; a < FIELD_LENGTH; a++) {
                    if (hb.minimums[a][0] != 0.0f || hb.maximums[a][0] != 1.0f)
                        clauseCount++;
                }
            }

            // get our accuracy now for this fold.
            map<pair<int, int>, PointSummary> pointSummaries;
            acc += testAccuracyOfHyperBlocks(hyperBlocks, testData, trainingData,pointSummaries, nearestNeighborK, similarityThreshold);
            blockCount += hyperBlocks.size();
            cCount += clauseCount;

            cout << "Block count: " << hyperBlocks.size() << endl;

        } // end of one train/test loop
        else {
            vector<vector<HyperBlock>> oneToOneBlocks = oneToOneHyper(trainingData, eachClassBestVectorIndex, classPairs);
            evaluateOneToOneHyperBlocks(oneToOneBlocks, testData, classPairs, NUM_CLASSES);
        }

    } // end of one train/test loop

    float avgAcc = float (acc) / float(k);
    float blockAvg = float(blockCount) / float(k);
    float clauseAvg = float(cCount) / float(k);

    if (hidePrinting)
        cout.rdbuf(oldBuf);           // back to console

    cout << "OVERALL ACCURACY " << avgAcc << endl;
    cout << "Average block count " << blockAvg << endl;
    cout << "Average clause count " << clauseAvg << endl;

    return {avgAcc, blockAvg, clauseAvg};
}

vector<float> runKFoldWithLevelNBlocks(vector<vector<vector<float>>> &dataset, bool takeUserInput = false, int removalCount = 0, int nearestNeighborK = 5, float similarityThreshold = 0.25f, bool hidePrinting = false, const int HB_LEVEL = 2) {

    if (dataset.empty()) {
        cout << "Please enter a training dataset before using K Fold validation" << endl;
        return {-1, -1, -1};
    }

    int k;
    // if we're taking input run it like normal. using this variable lets us just do it this way.
    if (takeUserInput) {
        cout << "Please Enter a K value:\t";
        cin >> k;

        // Clear the newline from the input buffer.
        cin.ignore(numeric_limits<streamsize>::max(), '\n');

        if (cin.fail() || k < 2) {
            cout << "Error: Invalid input. Please enter a valid integer greater than 1." << endl;
            // Clear the error state and ignore any remaining input.
            cin.clear();
            cin.ignore(numeric_limits<streamsize>::max(), '\n');
            return {-1, -1, -1};
        }
    }
    // if we're not using user input, we are testing for best accuracy and we can use 10.
    else
        k = 10;


    // used to hide the printing of the regular kFold testing stuff. so that when we are finding best parameters we don't have all that printing
    streambuf* oldBuf = nullptr;
    if (hidePrinting) {
        ostringstream nullSink;
        oldBuf = cout.rdbuf(nullSink.rdbuf());   // silence everything
    }

    vector<vector<vector<vector<float>>>> kFolds = DataUtil::splitDataset(dataset, k);
    // stats trackers for cross folds.
    float acc = 0.0f;
    int blockCount = 0;
    int cCount = 0;

    // generate blocks with a training set which is all folds except i. using i as the test dataset.
    for (int i = 0; i < k; i++) {

        // trainingData will store all folds except the i-th as training data.
        vector<vector<vector<float>>> trainingData(NUM_CLASSES);

        // Loop through all folds except i and accumulate points by class.
        for (int fold = 0; fold < k; fold++) {
            if (fold == i) continue; // skip test fold

            // build our training data
            for (int cls = 0; cls < NUM_CLASSES; cls++) {
                // Append all points from kFolds[fold][cls] to trainingData[cls]
                trainingData[cls].insert(trainingData[cls].end(), kFolds[fold][cls].begin(), kFolds[fold][cls].end());
            }
        }

        // little thing. causes issues if we don't reset when we make a new training set
        Knn::deviationsComputed = false;

        // The test dataset for this iteration is simply fold i.
        vector<vector<vector<float>>> testData = kFolds[i];

        // now that our data is set up with training and testing, we simply do business as usual. we are going to do our LDA on the train data, then just do our block generation and simplification
        // Run LDA on the training data.
        vector<vector<float>>bestVectors;
        // Initialize indexes for each class.
        vector<vector<int>> bestVectorsIndexes = vector<vector<int> >(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
        vector<int> eachClassBestVectorIndex = vector<int>(NUM_CLASSES);
        computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);

        cout << "----------------------------FOLD " << (i + 1) << " RESULTS----------------------------------" << endl;
        vector<HyperBlock> hyperBlocks;

        // ------------------------------------------
        // GENERATING BLOCKS BUSINESS AS USUAL
        IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);

        // now we iteratively increase the level of the blocks to whatever level
        vector<vector<vector<float>>> levelNData = trainingData;

        for (int level = 1; level < HB_LEVEL; level++) {
            vector<HyperBlock> thisLevelBlocks;
            // make our new set of blocks, and save this set of envelope cases. now we can reduce the training set iteratively.
            levelNData = move(IntervalHyperBlock::generateNextLevelHBs(levelNData, hyperBlocks, thisLevelBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS));

            // updating the train data itself actually allows us to perform better. we shrink the training set, and the KNN does better in this way.
            // trainingData = move(IntervalHyperBlock::generateNextLevelHBs(trainingData, hyperBlocks, thisLevelBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS));

            hyperBlocks  = move(thisLevelBlocks);   // advance to new level
        }

        // simplify them, with the simplification count we have specifed as a parameter. usually 0, but playing with this value can get us better results because we are removing more blocks
        Simplifications::REMOVAL_COUNT = removalCount;
        Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);

        int totalPoints = 0;
        for (const auto &c : trainingData)
            totalPoints += c.size();

        // clause count computed here because sometimes we don't simplify
        int clauseCount = 0;
        for (const auto &hb : hyperBlocks) {
            for (int a = 0; a < FIELD_LENGTH; a++) {
                if (hb.minimums[a][0] != 0.0f || hb.maximums[a][0] != 1.0f)
                    clauseCount++;
            }
        }

        // get our accuracy now for this fold.
        map<pair<int, int>, PointSummary> pointSummaries;
        acc += testAccuracyOfHyperBlocks(hyperBlocks, testData, trainingData,pointSummaries, nearestNeighborK, similarityThreshold);
        blockCount += hyperBlocks.size();
        cCount += clauseCount;
    } // end of one train/test loop

    float avgAcc = float (acc) / float(k);
    float blockAvg = float(blockCount) / float(k);
    float clauseAvg = float(cCount) / float(k);

    if (hidePrinting)
        cout.rdbuf(oldBuf);           // back to console if we had printing disabled.

    cout << "OVERALL ACCURACY " << avgAcc << endl;
    cout << "Average block count " << blockAvg << endl;
    cout << "Average clause count " << clauseAvg << endl;

    return {avgAcc, blockAvg, clauseAvg};

}

float evaluateOneToSomeHBs(const vector<vector<HyperBlock>>& oneToSomeBlocks, const vector<vector<vector<float>>>& testData) {
    vector<vector<long>> confusionMatrix(NUM_CLASSES, vector<long>(NUM_CLASSES, 0));
    int totalPoints = 0;
    for(const auto& c : testData) {
        totalPoints += c.size();
    }

    int incorrect = 0;
    int correct = 0;
    int pointsTested = 0;
    // Go through the classes
    for(int i = 0; i < NUM_CLASSES; i++) {
        for(int j = 0; j < testData[i].size(); j++) {
            pointsTested++;
            const auto& point = testData[i][j];
            bool classified = false;

            for(int sieveLvl = 0; sieveLvl < oneToSomeBlocks.size(); sieveLvl++) {
                for(const auto& hb : oneToSomeBlocks[sieveLvl]) {
                    if(hb.inside_HB(point.size(), point.data())) {
                        classified = true;

                        // If it is of the wrong class.
                        if(hb.classNum != i) {
                            incorrect++;
                        }
                        else {
                            correct++;
                        }

                        confusionMatrix[i][hb.classNum]++;
                        break;
                    }
                }
                if (classified) break; // do not keep checking once classified
            }
        }
    }

    for (const auto& row : confusionMatrix) {
        for (int i = 0; i < row.size(); i++) {
            cout << setw(5) << row[i] << " ";
        }
        cout << endl;
    }

    PrintingUtil::printConfusionMatrix(confusionMatrix, NUM_CLASSES, CLASS_MAP_INT);

    // Calculate the coverage.
    float coverage = static_cast<float>(incorrect + correct) / static_cast<float>(totalPoints);
    float acc = static_cast<float>(correct) / static_cast<float>(correct + incorrect);

    cout << "Coverage %: " << coverage * 100.0f << "%" << endl;
    cout << "Coverage total: " << (incorrect + correct) << " out of " << totalPoints << endl;
    cout << "Overall Accuracy: " << acc * 100.0f << "%" << endl;
    cout << "Total number of points tested: " << pointsTested << endl;
    return acc;
}

// -------------------------------------------------------------------------
// Asynchronous mode: run when argc >= 2
int runAsync(int argc, char* argv[]) {
    // Local variables for async mode
    string normalizedSaveFile;
    string hyperBlocksImportFileName;
    string trainingDataFileName;
    string testingDataFileName;
    string hyperBlocksExportFileName;

    // 3-D datasets
    vector<vector<vector<float>>> testData;
    vector<vector<vector<float>>> trainingData;

    // Normalization vectors (will be resized later)
    vector<float> minValues;
    vector<float> maxValues;

    // Store our HyperBlocks
    vector<HyperBlock> hyperBlocks;

    // Ultra confusion matrix
    vector<vector<long>> ultraConfusionMatrix;

    // Variables to be set by LDA

    if (argc > 3) {
        cout << "TOO MANY ARGUMENTS!" << endl;
        exit(1);
    }

    if (argc == 3) {
        // Set a global or externally-declared variable
        COMMAND_LINE_ARGS_CLASS = stoi(argv[2]);
        cout << "Running on class index " << COMMAND_LINE_ARGS_CLASS << endl;
    }

    // Process training data from file provided as first argument
    trainingData = DataUtil::dataSetup(argv[1], CLASS_MAP, CLASS_MAP_INT);
    cout << "NUM ATTRIBUTES : " << FIELD_LENGTH << endl;
    cout << "NUM CLASSES : " << NUM_CLASSES << endl;

    // Resize normalization vectors based on FIELD_LENGTH
    minValues.assign(FIELD_LENGTH, numeric_limits<float>::infinity());
    maxValues.assign(FIELD_LENGTH, -numeric_limits<float>::infinity());

    DataUtil::findMinMaxValuesInDataset(trainingData, minValues, maxValues, FIELD_LENGTH);
    DataUtil::minMaxNormalization(trainingData, minValues, maxValues, FIELD_LENGTH);

    cout << "RUNNING LDA" << endl;
    // Run LDA on the training data.
    vector<vector<float>>bestVectors;
    // Initialize indexes for each class
    vector<vector<int>> bestVectorsIndexes = vector<vector<int> >(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
    vector<int> eachClassBestVectorIndex = vector<int>(NUM_CLASSES);
    computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);

    IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
    cout << "HYPERBLOCK GENERATION FINISHED!" << endl;
    cout << "WE FOUND " << hyperBlocks.size() << " HYPERBLOCKS!" << endl;
    cout << "BEGINNING SIMPLIFICATIONS" << endl;

    string nonSimplified = string("NonSimplifiedBlocksClass") + to_string(COMMAND_LINE_ARGS_CLASS);
    DataUtil::saveBasicHBsToCSV(hyperBlocks, nonSimplified, FIELD_LENGTH);

    vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);
    int totalPoints = 0;
    for (const auto &c : trainingData)
        totalPoints += c.size();
    cout << "After removing useless blocks we have: " << result[1] << " clauses\n";
    cout << "Ran simplifications: " << result[0] << " Times" << endl;
    cout << "We had: " << totalPoints << " points\n";

    string simplified = string("SimplifiedBlocks") + to_string(COMMAND_LINE_ARGS_CLASS);
    DataUtil::saveBasicHBsToCSV(hyperBlocks, simplified, FIELD_LENGTH);
    return 0;
}


void evaluateOneToOneHyperBlocks(
    const vector<vector<HyperBlock>>& oneToOneHBs,
    const vector<vector<vector<float>>>& testSet,
    const vector<pair<int, int>>& classPairs,
    int numClasses
) {
    int totalPoints = 0;
    int correctPoints = 0;

    vector<vector<int>> confusion(numClasses, vector<int>(numClasses, 0));
    vector<int> correctPerClass(numClasses, 0);
    vector<int> totalPerClass(numClasses, 0);


    for (int actualClass = 0; actualClass < numClasses; ++actualClass) {
        for (const auto& point : testSet[actualClass]) {
            vector<int> votes(numClasses, 0);

            // Run all pairwise comparisons
            for (int i = 0; i < numClasses; ++i) {
                for (int j = i + 1; j < numClasses; ++j) {
                    // Find the correct index using classPairs
                    int findPairIndex = -1;
                    for (int idx = 0; idx < classPairs.size(); ++idx) {
                        if ((classPairs[idx].first == i && classPairs[idx].second == j) ||
                            (classPairs[idx].first == j && classPairs[idx].second == i)) {
                            findPairIndex = idx;
                            break;
                            }
                    }
                    if (findPairIndex == -1) {
                        cerr << "Error: could not find block set for classes " << i << " and " << j << "\n";
                        continue;
                    }

                    const auto& pairHBs = oneToOneHBs[findPairIndex];


                    for (const auto& block : pairHBs) {
                        if (block.classNum == i && block.inside_HB(point.size(), point.data())) {
                            votes[i]++;
                        }
                        if (block.classNum == j && block.inside_HB(point.size(), point.data())) {
                            votes[j]++;
                        }
                    }
                }
            }

            // Majority vote
            int predictedClass = distance(votes.begin(), max_element(votes.begin(), votes.end()));
            if (votes[predictedClass] == 0) {
                // No vote: optionally handle as unclassified
                continue;
            }

            confusion[actualClass][predictedClass]++;
            totalPerClass[actualClass]++;
            if (predictedClass == actualClass) {
                correctPerClass[actualClass]++;
                correctPoints++;
            }
            totalPoints++;
        }
    }

    // Output
    cout << "\nConfusion Matrix:\n";
    for (int i = 0; i < numClasses; ++i) {
        for (int j = 0; j < numClasses; ++j) {
            cout << confusion[i][j] << "\t";
        }
        cout << "\n";
    }

    cout << "\nPer-Class Accuracy:\n";
    for (int i = 0; i < numClasses; ++i) {
        float acc = totalPerClass[i] ? static_cast<float>(correctPerClass[i]) / totalPerClass[i] : 0.0f;
        cout << "Class " << i << ": " << acc * 100.0f << "%\n";
    }

    cout << "\nOverall Accuracy: " << (static_cast<float>(correctPoints) / totalPoints) * 100.0f << "%\n";
}



/**
 * This function is a experimental one to test how weighting the votes of HBs by their performance
 * on a validation set. At a high level what we do in this is:
 *
 * 1. Split the training dataset into "reducedTraining" and "validationData"
 * 2. Generate HBs using the "reducedTraining" dataset.
 * 3. Evaluate the HBs using the "validationData" as a testing set. Within this we build PointSummaries
 *    which links every block index to the points from validation which fell inside the block. This allows
 *    for metrics like precision to be kept. Specifically, each block stores, its precision AND the amount of precision
 *    that each other class of points caused it to lose. The latter gives us a prediction of what the block might misclassify
 *    as belonging to its own class. Ex. high loss of precision from class y, means this blocks might actually vote for a y.
 *
 * 4. We save the afforementioned stats, then evaluate the HBs on the TRUE TESTING dataset using the precision weighted voting.
 * 5. Return the accuracy obtained.
 *
 *
 * @param trainingData  The training dataset input by the user.
 * @param eachClassBestVectorIndex Will be NUM_CLASSES long. Stores the best attribute to sort each class by during intervalHyper generation process.
 *                                 basically the most separating attribute for the specific class.
 * @param hyperBlocks              Empty Hyperblocks array, will be full when the function exits.
 * @param testingData      The dataset to be used during the final testing phase of the precision weighted hbs.
 * @param bestVectorsIndexes The order of attributes for each class. ex: {{0,1,2}, {2, 1, 0}} the attribute removal order
 *                           when simplifications are run for class 1 would be 0,1,2. The second class would be 2,1,0
 * @return  The accuracy, could be the stat struct if confusion matrix function is updated.
 */
float genAndRunPrecisionWeightedHBs(vector<vector<vector<float>>>& trainingData,
                                    vector<int> eachClassBestVectorIndex, vector<HyperBlock>& hyperBlocks,
                                    vector<vector<vector<float>>>& testingData,
                                    vector<vector<int>> bestVectorsIndexes
) {

    map<pair<int, int>, PointSummary> pointSummaries;

    //TODO: We need to set up a temp so that the training data is reset to its entire version after running this program.
    vector<vector<vector<float>>> validationData;
    DataUtil::createValidationSplit( trainingData, validationData, .10, 42);

    // Build the hbs
    IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
    Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);

    // Test the validation HBS, returns confusion matrix, vector<vector<long>>
    vector<vector<vector<float>>> stillUnclassified(NUM_CLASSES);
    vector<vector<long>> confusionMatrix = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, validationData, ClassificationTests::HYPERBLOCKS,stillUnclassified , NUM_CLASSES, pointSummaries);

    for(auto& hb : hyperBlocks) {
        hb.setHBPrecisions(pointSummaries, NUM_CLASSES);
    }

    // Go through and make a non-distinct confusion matrix.
    std::vector<std::vector<long>> ultraConfusionMatrix(NUM_CLASSES, std::vector<long>(NUM_CLASSES, 0));

    for (const auto& entry : pointSummaries) {
        const PointSummary& summary = entry.second;
        int trueClass = summary.classIdx;

        for (const BlockInfo& hit : summary.blockHits) {
            int blockClass = hit.blockClass;
            ultraConfusionMatrix[trueClass][blockClass] += 1;
        }
    }


    pointSummaries.clear();
    vector<vector<long>> newConfusion = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, testingData, ClassificationTests::PRECISION_WEIGHTED, stillUnclassified , NUM_CLASSES, pointSummaries);
    cout << "\nPrecision Weighted Matrix " << endl;
    PrintingUtil::printConfusionMatrix(newConfusion, NUM_CLASSES, CLASS_MAP_INT);

    vector<vector<vector<float>>> unclassed(NUM_CLASSES);
    vector<vector<long>> knnMatrix = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, stillUnclassified, ClassificationTests::PURE_KNN, unclassed , NUM_CLASSES, pointSummaries);
    cout << "\nKNN matrix" << endl;
    PrintingUtil::printConfusionMatrix(knnMatrix, NUM_CLASSES, CLASS_MAP_INT);

    for(int i = 0; i < knnMatrix.size(); ++i) {
        for (int j = 0; j < knnMatrix[i].size(); ++j) {
            newConfusion[i][j] += knnMatrix[i][j];
        }
    }
    cout << "\nOld Matrix" << endl;
    vector<vector<long>> oldConf = ClassificationTests::buildConfusionMatrix(hyperBlocks, trainingData, testingData, ClassificationTests::HYPERBLOCKS, stillUnclassified , NUM_CLASSES, pointSummaries);
    PrintingUtil::printConfusionMatrix(oldConf, NUM_CLASSES, CLASS_MAP_INT);

    return PrintingUtil::printConfusionMatrix(newConfusion, NUM_CLASSES, CLASS_MAP_INT);
}



vector<float> precisionKFold(vector<vector<vector<float>>> &dataset, int nearestNeighborK = 5, float similarityThreshold = 0.25f, bool hidePrinting = false) {
    if (dataset.empty()) {
        cout << "Please enter a training dataset before using K Fold validation" << endl;
        return {-1, -1, -1};
    }

    int k = 10;
    streambuf* oldBuf = nullptr;
    if (hidePrinting) {
        ostringstream nullSink;
        oldBuf = cout.rdbuf(nullSink.rdbuf());
    }

    vector<vector<vector<vector<float>>>> kFolds = DataUtil::splitDataset(dataset, k);
    float acc = 0.0f;
    int blockCount = 0;
    int cCount = 0;

    for (int i = 0; i < k; i++) {
        vector<vector<vector<float>>> trainingData(NUM_CLASSES);

        for (int fold = 0; fold < k; fold++) {
            if (fold == i) continue;
            for (int cls = 0; cls < NUM_CLASSES; cls++) {
                trainingData[cls].insert(trainingData[cls].end(), kFolds[fold][cls].begin(), kFolds[fold][cls].end());
            }
        }

        vector<vector<vector<float>>> testData = kFolds[i];

        vector<vector<float>> bestVectors;
        vector<vector<int>> bestVectorsIndexes(NUM_CLASSES, vector<int>(FIELD_LENGTH, 0));
        vector<int> eachClassBestVectorIndex(NUM_CLASSES);
        computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);

        cout << "----------------------------FOLD " << (i + 1) << " RESULTS----------------------------------" << endl;

        vector<HyperBlock> hyperBlocks;

        acc += genAndRunPrecisionWeightedHBs(trainingData, eachClassBestVectorIndex, hyperBlocks, testData, bestVectorsIndexes);

        int clauseCount = 0;
        for (const auto &hb : hyperBlocks) {
            for (int a = 0; a < FIELD_LENGTH; a++) {
                if (hb.minimums[a][0] != 0.0f || hb.maximums[a][0] != 1.0f)
                    clauseCount++;
            }
        }

        blockCount += hyperBlocks.size();
        cCount += clauseCount;

        cout << "Block count: " << hyperBlocks.size() << endl;
    }

    float avgAcc = acc / k;
    float blockAvg = static_cast<float>(blockCount) / k;
    float clauseAvg = static_cast<float>(cCount) / k;

    if (hidePrinting)
        cout.rdbuf(oldBuf);

    cout << "OVERALL ACCURACY " << avgAcc << endl;
    cout << "Average block count " << blockAvg << endl;
    cout << "Average clause count " << clauseAvg << endl;

    return {avgAcc, blockAvg, clauseAvg};
}


// -------------------------------------------------------------------------
// Interactive mode: run when argc < 2
void runInteractive() {
    // Local variables for interactive mode
    string normalizedSaveFile;
    string hyperBlocksImportFileName;
    string trainingDataFileName;
    string testingDataFileName;
    string hyperBlocksExportFileName;

    vector<vector<vector<float>>> testData;
    vector<vector<vector<float>>> trainingData;
    vector<vector<vector<float>>> validationData;

    vector<float> minValues;
    vector<float> maxValues;

    vector<HyperBlock> hyperBlocks;

    vector<vector<long>> ultraConfusionMatrix;

    vector<vector<float>> bestVectors;
    vector<vector<int>> bestVectorsIndexes;
    vector<int> eachClassBestVectorIndex;

    // Class ordering
    vector<int> order;
    vector<vector<HyperBlock>> oneToOneBlocks;
    vector<pair<int, int>> classPairsOut;

    vector<vector<HyperBlock>> oneToRestBlocks;

    int normChoice;

    bool running = true;
    int choice;
    while (running) {
        PrintingUtil::displayMainMenu();
        cin >> choice;
        cin.clear();
        cin.ignore(numeric_limits<streamsize>::max(), '\n');

        switch (choice) {
            case 1: { // IMPORT TRAINING DATA
                cout << "Enter training data filename: " << endl;
                #ifdef _WIN32
                    system("dir datasets");
                #else
                    system("ls datasets");
                #endif
                getline(cin, trainingDataFileName);
                string fullPath = "datasets" + string(PATH_SEPARATOR) + trainingDataFileName;
                CLASS_MAP_INT.clear();
                trainingData = DataUtil::dataSetup(fullPath.c_str(), CLASS_MAP, CLASS_MAP_INT);

                cout << "Choose normalization method:\n";
                cout << "  1. Min-Max normalize using dataset bounds\n";
                cout << "  2. Normalize by fixed max value (e.g., 255)\n";
                cout << "  3. No normalization\n";
                cout << "Enter choice (1-3): ";
                cin >> normChoice;
                cin.ignore();  // flush newline


                if (normChoice == 1) {
                    minValues.assign(FIELD_LENGTH, numeric_limits<float>::infinity());
                    maxValues.assign(FIELD_LENGTH, -numeric_limits<float>::infinity());
                    DataUtil::findMinMaxValuesInDataset(trainingData, minValues, maxValues, FIELD_LENGTH);
                    DataUtil::minMaxNormalization(trainingData, minValues, maxValues, FIELD_LENGTH);

                    order = computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);
                } else if (normChoice == 2) {
                    float fixedMax;
                    cout << "Enter fixed max value (e.g., 255): ";
                    cin >> fixedMax;
                    cin.ignore();  // flush newline

                    minValues.assign(FIELD_LENGTH, 0.0f);
                    maxValues.assign(FIELD_LENGTH, fixedMax);

                    DataUtil::minMaxNormalization(trainingData, minValues, maxValues, FIELD_LENGTH);
                    order = computeLDAOrdering(trainingData, bestVectors, bestVectorsIndexes, eachClassBestVectorIndex);
                } else {
                    cout << "Skipping normalization.\n";
                }

                // this has to get set false each time we make a new dataset or else we are going to make a seg fault when we test a second dataset.
                Knn::deviationsComputed = false;

                PrintingUtil::waitForEnter();
                break;
            }
            case 2: { // IMPORT TESTING DATA
                cout << "Enter testing data filename: " << endl;
                system("ls datasets");
                getline(cin, testingDataFileName);
                string fullPath = "datasets" + string(PATH_SEPARATOR) + testingDataFileName;

                // clear these two maps to prevent issues when using a second test set.
                CLASS_MAP_TESTING.clear();
                CLASS_MAP_TESTING_INT.clear();

                testData = DataUtil::dataSetup(fullPath, CLASS_MAP_TESTING, CLASS_MAP_TESTING_INT);

                if (normChoice == 1 || normChoice == 2) {
                    DataUtil::normalizeTestSet(testData, minValues, maxValues, FIELD_LENGTH);
                } else {
                    cout << "Skipping normalization.\n";
                }

                // Normalize and reorder testing data as needed.
                testData = DataUtil::reorderTestingDataset(testData, CLASS_MAP, CLASS_MAP_TESTING);

                for(const auto& cls: testData) {
                    cout << cls.size() << endl;
                }
                PrintingUtil::waitForEnter();
                break;
            }
            case 3: { // SAVE NORMALIZED TRAINING DATA
                cout << "Enter the file to save the normalized training data to: " << endl;
                getline(cin, normalizedSaveFile);
                DataUtil::saveNormalizedVersionToCsv(normalizedSaveFile, trainingData);
                cout << "Saved normalized training data to: " << normalizedSaveFile << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 4: { // IMPORT EXISTING HYPERBLOCKS
                cout << "Enter existing hyperblocks file name: " << endl;
                getline(cin, hyperBlocksImportFileName);
                hyperBlocks = DataUtil::loadBasicHBsFromBinary(hyperBlocksImportFileName);

                cout << "HyperBlocks imported from file " << hyperBlocksImportFileName << " successfully" << endl;

                for(HyperBlock& hb: hyperBlocks){
                  hb.find_avg_and_size(trainingData);
                }

                PrintingUtil::waitForEnter();
                break;

            }
            case 5: { // EXPORT HYPERBLOCKS
                cout << "Enter the file to save HyperBlocks to: " << endl;
                getline(cin, hyperBlocksExportFileName);
                DataUtil::saveBasicHBsToBinary(hyperBlocks, hyperBlocksExportFileName, FIELD_LENGTH);
                break;
            }
            case 6: { // GENERATE NEW HYPERBLOCKS
                if (trainingData.empty()) {
                    cout << "\nError: Please import training data first." << endl;
                    PrintingUtil::waitForEnter();
                } else {
                    hyperBlocks.clear();
                    IntervalHyperBlock::generateHBs(trainingData, hyperBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS);
                }

                cout << "Finished Generating HyperBlocks" << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 7: {       // SIMPLIFY HYPERBLOCKS
             
                // testing for time and to determine if they are doing the same.
                auto start = chrono::high_resolution_clock::now();
                vector<HyperBlock> simpleVersionBlocks = hyperBlocks;
                Simplifications::removeUselessAttrNoDisjunction(simpleVersionBlocks, trainingData, bestVectorsIndexes);
                auto end = chrono::high_resolution_clock::now();
                chrono::duration<double> diff = end - start;
                cout << "Time taken (new version): " << diff.count() << " seconds" << endl;


                start = chrono::high_resolution_clock::now();
                Simplifications::removeUselessAttr(hyperBlocks, trainingData, bestVectorsIndexes);
                end = chrono::high_resolution_clock::now();
                diff = end - start;
                cout << "Time taken (old version): " << diff.count() << " seconds" << endl;

                int newClauseCount = 0;
                for (const auto &hb : simpleVersionBlocks) {
                    for (int a = 0; a < FIELD_LENGTH; a++) {
                        if (hb.minimums[a][0] != 0.0f || hb.maximums[a][0] != 1.0f)
                            newClauseCount++;
                    }
                }

                int oldClauseCount = 0;
                for (const auto &hb : hyperBlocks) {
                    for (int a = 0; a < FIELD_LENGTH; a++) {
                        if (hb.minimums[a][0] != 0.0f || hb.maximums[a][0] != 1.0f)
                            oldClauseCount++;
                    }
                }


                cout << "New clause count: " << newClauseCount << endl;
                cout << "Old clause count: " << oldClauseCount << endl;

                /*
                vector<int> result = Simplifications::runSimplifications(hyperBlocks, trainingData, bestVectorsIndexes);
                

                int totalPoints = 0;

                for (const auto &c : trainingData) totalPoints += c.size();

                cout << "After removing useless blocks we have: " << newClauseCount << " clauses\n";
                cout << "We got a final total of: " << hyperBlocks.size() << " blocks." << endl;
                cout << "We had: " << totalPoints << " points of training data\n";
                for (const auto &c : trainingData) totalPoints += c.size();

                cout << "After removing useless blocks we have: " << result[1] << " clauses\n";
                cout << "We got a final total of: " << hyperBlocks.size() << " blocks." << endl;
                cout << "We had: " << totalPoints << " points of training data\n";
                */
                PrintingUtil::waitForEnter();
                break;
            }
            case 8: { // TEST HBs ON CURRENT TESTING DATASET
                cout << "Testing HBs on testing dataset" << endl;
                map<pair<int, int>, PointSummary> pointSummaries;
                testAccuracyOfHyperBlocks(hyperBlocks, testData, trainingData, pointSummaries);
                PrintingUtil::waitForEnter();
                break;
            }
            case 9: {   // TEST 1-1 HBs ON CURRENT TESTING DATASET
                evaluateOneToOneHyperBlocks(oneToOneBlocks, testData, classPairsOut, NUM_CLASSES);
                PrintingUtil::waitForEnter();
                break;
            }
            case 10: {  // RUN K-FOLD CROSS VALIDATION
                vector<pair<int,int>> classPairs{}; // just needed because we have to pass in something

                // run the k fold, taking the user input for number of k. using default values for removal count, k and whatnot
                runKFold(trainingData, classPairs, true, true);  //precisionKFold(trainingData);
                PrintingUtil::waitForEnter();
                break;
            }
            case 11: {  // GENERATE 1-1 HBs
                if (trainingData.empty()) {
                    cout << "\nError: Please import training data first." << endl;
                    PrintingUtil::waitForEnter();
                } else {
                    oneToOneBlocks.clear();
                    oneToOneBlocks = oneToOneHyper(trainingData, eachClassBestVectorIndex, classPairsOut);
                }

                cout << "Finished Generating 1-1 HyperBlocks" << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 12: {    // IMPORT 1-1 HBs
                cout << "Enter 1-1 Hyperblocks file name: " << endl;
                getline(cin, hyperBlocksImportFileName);
                oneToOneBlocks = DataUtil::loadOneToOneHBsFromBinary(hyperBlocksImportFileName, classPairsOut);
                cout << "HyperBlocks imported from file " << hyperBlocksImportFileName << " successfully" << endl;

                PrintingUtil::waitForEnter();
                break;
            }
            case 13: { // EXPORT 1-1 HBs
                cout << "Enter the file to save Hyperblocks to: " << endl;
                getline(cin, hyperBlocksExportFileName);
                DataUtil::saveOneToOneHBsToBinary(oneToOneBlocks, hyperBlocksExportFileName);
                break;
            }
            case 14: {  // RUN K-FOLD USING THE 1-1 HBs
                vector<pair<int,int>> classPairs{}; // need to make up the class pairings

                runKFold(trainingData, classPairs, false, true);
                PrintingUtil::waitForEnter();
                break;
            }
            case 15: {  // GENERATE 1-Rest HBs
                // POINT BASED ORDER
                oneToRestBlocks.clear();


                auto start = chrono::high_resolution_clock::now();
                oneToRestBlocks = oneToRestHyper(trainingData, eachClassBestVectorIndex);
                auto end = chrono::high_resolution_clock::now();
                chrono::duration<double> diff = end - start;

                // Flatten the set of HBs by moving them.
                vector<HyperBlock> allRestBlocks;
                for (auto& blockSet : oneToRestBlocks) {
                    allRestBlocks.insert(allRestBlocks.end(),make_move_iterator(blockSet.begin()),make_move_iterator(blockSet.end()));
                }

                DataUtil::saveBasicHBsToBinary(allRestBlocks, "digitBlocksRest.csv", FIELD_LENGTH);
                cout << "Finished Generating one to Some blocks." << endl;
                cout << "Elapsed time: " << diff.count() << " seconds\n";
            }
            case 16: {

                int maxRemoval;
                int maxK;

                cout << "Please enter a max threshold size to test our blocks with " << endl;
                cin >> maxRemoval;
                // Clear the newline from the input buffer.
                cin.ignore(numeric_limits<streamsize>::max(), '\n');

                if (cin.fail() || maxRemoval < 0) {
                    cout << "Error: Invalid input. Please enter a valid integer greater than 1." << endl;
                    // Clear the error state and ignore any remaining input.
                    cin.clear();
                    cin.ignore(numeric_limits<streamsize>::max(), '\n');
                    return;
                }

                cout << "Please enter a max K value to test our KNN with " << endl;
                cin >> maxK;
                // Clear the newline from the input buffer.
                cin.ignore(numeric_limits<streamsize>::max(), '\n');

                cout << "What level HBs are we testing?" << endl;
                int blockLevel;
                cin >> blockLevel;
                cin.ignore(numeric_limits<streamsize>::max(), '\n');

                if (cin.fail() || maxK < 0) {
                    cout << "Error: Invalid input. Please enter a valid integer greater than 1." << endl;
                    // Clear the error state and ignore any remaining input.
                    cin.clear();
                    cin.ignore(numeric_limits<streamsize>::max(), '\n');
                    return;
                }

                vector<int> removalCounts;
                for (int i = 0; i <= maxRemoval; i++) {
                    removalCounts.push_back(i);
                }

                vector<int> kVals;
                for (int i = 1; i <= maxK; i += 2) {
                    kVals.push_back(i);
                }

                vector<float> thresholds{0.15, 0.2, 0.25, 0.3};

                // findBestParameters(trainingData, maxRemoval, maxK);
                findBestParameters(trainingData, kVals, thresholds, maxRemoval, false, blockLevel);
                PrintingUtil::waitForEnter();
                break;
            }
            case 17: {
                if (trainingData.empty()) {
                    cout << "\nError: Please import training data first." << endl;
                    PrintingUtil::waitForEnter();
                } else {
                    vector<HyperBlock> newBlocks;
                    trainingData = move(IntervalHyperBlock::generateNextLevelHBs(trainingData, hyperBlocks, newBlocks, eachClassBestVectorIndex, FIELD_LENGTH, COMMAND_LINE_ARGS_CLASS));
                    hyperBlocks = move(newBlocks);
                }
                static int levelN = 1;
                cout << "Finished Generating level " << ++levelN << " HBs" << endl;
                PrintingUtil::waitForEnter();
                break;
            }
            case 18: {
                // run our level N k fold function
                runKFoldWithLevelNBlocks(trainingData, false, 1, 1, .25f);
                PrintingUtil::waitForEnter();
                break;
            }
            case 19: {
                // MERGE FIX: This was going to be in 17, but level N hbs displaces it
                genAndRunPrecisionWeightedHBs(trainingData, eachClassBestVectorIndex, hyperBlocks , testData, bestVectorsIndexes);
                PrintingUtil::waitForEnter();
                break;
            }
            case 20: {
                running = false;
                break;
            }
            default: {
                cout << "\nInvalid choice. Please try again." << endl;
                PrintingUtil::waitForEnter();
                break;
            }
        }
    }
}

// -------------------------------------------------------------------------
// Main entry point: choose mode based on argc.
int main(int argc, char* argv[]) {

    // Command line input mode, allows you to specify in command line what to do
    if (argc >= 2)
        return runAsync(argc, argv);

    // Interactive input loop, options to import data, train, test, save, etc
    runInteractive();
    return 0;
}