//
// Created by Austin Snyder on 3/20/2025.
//
#include "Simplifications.h"
#include "../hyperblock_generation/MergerHyperBlock.cuh"
int Simplifications::REMOVAL_COUNT = 0;

/**
 * Runs our three kernel functions which remove useless blocks / Remove Redundant Blocks (R2A)
 *
 * Details are discussed further in "Fully Explainable Classification Models Using Hyperblocks", 2025. Ryan Gallagher, Austin Snyder, Boris Kovalerchuk
 */
void Simplifications::removeUselessBlocks(vector<vector<vector<float>>> &data, vector<HyperBlock>& hyper_blocks) {
    /*
     * The algorithm to remove useless blocks does basically this.
     *     - take one particular point in our dataset. Find the first HB that it fits into.
     *     - then, once everyone has found their first choice, we sum up the count of which HBs have how many points
     *     - then, we run it again, this time starting from the block which each point chose. We pick a new HB instead, if we find one which our point falls into, and which has a higher amount of points in it than our current
     *     - this is not a perfect way of doing it, but at least allows us to find the "most general blocks" based on the count of how many points are in each. This way we can then just delete whichever blocks we find with no *UNIQUE* points in them.
     *     * notice how we are putting all data in, and all blocks together. this allows us to find errors as well. we may find that a block is letting in wrong class points this way.
     */
    int FIELD_LENGTH = data[0][0].size();
    int datasetSize = 0;
    for (auto &c : data)
        datasetSize += c.size();

    vector<vector<float>> minMaxResult = DataUtil::flattenMinsMaxesForRUB(hyper_blocks, FIELD_LENGTH);
    vector<vector<float>> flattenedData =  DataUtil::flattenDataset(data);

    // Use references to avoid copying.
    const vector<float>& blockMins   = minMaxResult[0];
    const vector<float>& blockMaxes  = minMaxResult[1];

    // Cast each element from the third vector (floats) into ints.
    const vector<float> &edgesAsFloats = minMaxResult[2];
    vector<int> blockEdges;
    blockEdges.resize(minMaxResult[2].size());
    // cast result [2] to ints, since this is the block edges. the array which tells us where each block starts and ends (as indexes).
    transform(edgesAsFloats.begin(), edgesAsFloats.end(), blockEdges.begin(),
              [](float val) -> int { return static_cast<int>(val); });

    // Get the dataPointsArray (again using a reference).
    const vector<float>& dataPointsArray = flattenedData[0];

    const int numPoints = dataPointsArray.size() / FIELD_LENGTH;
    vector<int> dataPointBlocks(numPoints, 0);              // Each point's chosen block.
    const int numBlocks = hyper_blocks.size();                    // Number of hyperblocks.
    vector<int> numPointsInBlocks(numBlocks, 0);              // Count of points in each hyperblock.

    // Allocate device memory and copy data.
    float *d_dataPointsArray, *d_blockMins, *d_blockMaxes;
    int   *d_blockEdges;
    int *d_dataPointBlocks, *d_numPointsInBlocks;

    hipMalloc((void**)&d_dataPointsArray, sizeof(float) * numPoints * FIELD_LENGTH);
    hipMemcpy(d_dataPointsArray, dataPointsArray.data(), sizeof(float) * numPoints * FIELD_LENGTH, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_blockMins, sizeof(float) * blockMins.size());
    hipMemcpy(d_blockMins, blockMins.data(), sizeof(float) * blockMins.size(), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_blockMaxes, sizeof(float) * blockMaxes.size());
    hipMemcpy(d_blockMaxes, blockMaxes.data(), sizeof(float) * blockMaxes.size(), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_blockEdges, sizeof(int) * blockEdges.size());
    hipMemcpy(d_blockEdges, blockEdges.data(), sizeof(int) * blockEdges.size(), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_dataPointBlocks, sizeof(int) * numPoints);
    hipMemset(d_dataPointBlocks, 0, sizeof(int) * numPoints);

    hipMalloc((void**)&d_numPointsInBlocks, sizeof(int) * numBlocks);
    hipMemset(d_numPointsInBlocks, 0, sizeof(int) * numBlocks);

    // Determine grid and block sizes using CUDA occupancy.
    int minGridSize, blockSize;
    hipError_t err = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, assignPointsToBlocks, 0, 0);
    if (err != hipSuccess) {
        printf("CUDA error in hipOccupancyMaxPotentialBlockSize: %s\n", hipGetErrorString(err));
        exit(-1);
    }
    int gridSize = (numPoints + blockSize - 1) / blockSize;

    assignPointsToBlocksWrapper(d_dataPointsArray, FIELD_LENGTH, numPoints, d_blockMins, d_blockMaxes, d_blockEdges, numBlocks, d_dataPointBlocks, gridSize, blockSize);
    hipDeviceSynchronize();

    sumPointsPerBlockWrapper(d_dataPointBlocks, numPoints, d_numPointsInBlocks, gridSize, blockSize);
    hipDeviceSynchronize();

    findBetterBlocksWrapper(d_dataPointsArray, FIELD_LENGTH, numPoints, d_blockMins, d_blockMaxes, d_blockEdges, numBlocks, d_dataPointBlocks, d_numPointsInBlocks, gridSize, blockSize);
    hipDeviceSynchronize();

    // Reset the numPointsInBlocks array on the device, this is because we have now found better homes, and we are ready to recompute the sums.
    hipMemset(d_numPointsInBlocks, 0, sizeof(int) * numBlocks);
    sumPointsPerBlockWrapper(d_dataPointBlocks, numPoints, d_numPointsInBlocks, gridSize, blockSize);
    hipDeviceSynchronize();

    // Copy back the computed numPointsInBlocks.
    hipMemcpy(numPointsInBlocks.data(), d_numPointsInBlocks, sizeof(int) * numBlocks, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree((void *)d_dataPointsArray);
    hipFree((void *)d_blockMins);
    hipFree((void *)d_blockMaxes);
    hipFree((void *)d_blockEdges);
    hipFree((void *)d_dataPointBlocks);
    hipFree((void *)d_numPointsInBlocks);

    // Remove blocks with less than our count of unique points
    // unique points refers to the amount of points which are classified uniquely by this particular block.
    for (int i = numPointsInBlocks.size() - 1; i >= 0; i--) {
        if (numPointsInBlocks[i] <= REMOVAL_COUNT)
            hyper_blocks.erase(hyper_blocks.begin() + i);
    }
}

/**
 * Attempts to remove redundant attributes from hyperblocks.
 *
 * This can be done if the attribute is not needed for classification,
 * for example if everything is seperable by x1, then we don't need to keep x2.
 *
 * Details are discussed further in "Fully Explainable Classification Models Using Hyperblocks", 2025. Ryan Gallagher, Austin Snyder, Boris Kovalerchuk
 * @param hyper_blocks
 * @param data
 * @param attributeOrderings
 */
void Simplifications::removeUselessAttrNoDisjunction(vector<HyperBlock>& hyper_blocks, vector<vector<vector<float>>>& data, vector<vector<int>>& attributeOrderings) {
    const int FIELD_LENGTH = data[0][0].size();

    // Prepare host data by flattening your data structures.
    auto fMinMaxResult =  DataUtil::flatMinMaxNoEncode(hyper_blocks, FIELD_LENGTH);
    auto fDataResult =  DataUtil::flattenDataset(data);

    // Build host arrays from the flattened results:
    vector<float> mins = fMinMaxResult[0];
    vector<float> maxes = fMinMaxResult[1];

    const int numBlocks = static_cast<int>(hyper_blocks.size());

    vector<int> blockClasses(fMinMaxResult[3].size());
    for (size_t i = 0; i < fMinMaxResult[3].size(); i++) {
        blockClasses[i] = static_cast<int>(fMinMaxResult[3][i]);
    }

    // Prepare the dataset.
    int numPoints = static_cast<int>(fDataResult[0].size() / FIELD_LENGTH);
    vector<float> transposedData(fDataResult[0].size(), 0.0f);

    vector<int> classBorder(fDataResult[1].size());
    for (size_t i = 0; i < fDataResult[1].size(); i++) {
        classBorder[i] = static_cast<int>(fDataResult[1][i]);
    }
    int numClasses = static_cast<int>(hyper_blocks.size());

    std::vector<int> attributeOrderingsFlattened(attributeOrderings.size() * FIELD_LENGTH, 0);
    for (int i = 0; i < attributeOrderings.size(); i++) {
        copy(attributeOrderings[i].begin(), attributeOrderings[i].end(),
            attributeOrderingsFlattened.begin() + i * FIELD_LENGTH);
    }

    // Transpose the dataset, from point being a row, to a point being a column
    int rows = numPoints;
    int cols = FIELD_LENGTH;
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            transposedData[j*rows+i] = fDataResult[0][i * cols + j];
        }
    }

    // Device pointers.
    float* d_mins = nullptr;
    float* d_maxes = nullptr;
    int* d_blockClasses = nullptr;
    float* d_dataset = nullptr;
    int* d_classBorder = nullptr;
    int* d_attributeOrderingsFlattened = nullptr;

    // Allocate device memory.
    hipMalloc((void**)&d_mins, mins.size() * sizeof(float));
    hipMalloc((void**)&d_maxes, maxes.size() * sizeof(float));
    hipMalloc((void**)&d_blockClasses, blockClasses.size() * sizeof(int));
    hipMalloc((void**)&d_dataset, transposedData.size() * sizeof(float));
    hipMalloc((void**)&d_classBorder, classBorder.size() * sizeof(int));
    hipMalloc((void**)&d_attributeOrderingsFlattened, attributeOrderingsFlattened.size() * sizeof(int));

    // Copy host data to device.
    hipMemcpy(d_mins, mins.data(), mins.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_maxes, maxes.data(), maxes.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_blockClasses, blockClasses.data(), blockClasses.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dataset, transposedData.data(), transposedData.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_classBorder, classBorder.data(), classBorder.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_attributeOrderingsFlattened, attributeOrderingsFlattened.data(), attributeOrderingsFlattened.size() * sizeof(int), hipMemcpyHostToDevice);

    // Determine execution configuration.
    int blockSize;
    int gridSize;

    hipError_t err = hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, mergerHyperBlocks, 0, 0);
    gridSize = (numBlocks + blockSize - 1) / blockSize;

    // Launch the kernel.
    removeUselessAttributesNoDisjunctionsWrapper(d_mins, d_maxes, numBlocks, FIELD_LENGTH, d_blockClasses, d_dataset, numPoints, d_classBorder, numClasses, d_attributeOrderingsFlattened, gridSize, blockSize);
    //removeUselessAttributesNoDisjunctions(float *mins, float *maxes, const int numBlocks, const int FIELD_LENGTH, const int *blockClasses, const float *dataset, const int numPoints, const int *classBorder, const int numClasses, const int *attributeOrder){

    hipDeviceSynchronize();

    // Copy results from device  back to host.
    hipMemcpy(mins.data(), d_mins, mins.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(maxes.data(), d_maxes, maxes.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_mins);
    hipFree(d_maxes);
    hipFree(d_blockClasses);
    hipFree(d_dataset);
    hipFree(d_classBorder);
    hipFree(d_attributeOrderingsFlattened);

    // Go through the blocks, copy data back in.
    int index = 0;
    for(int i = 0; i < numBlocks; i++) {
        auto& hb = hyper_blocks[i];
        // Go through the attributes
        for(int attr = 0; attr < FIELD_LENGTH; attr++) {
            hb.minimums[attr][0] = mins[index];
            hb.maximums[attr][0] = maxes[index];
            index++;
        }
    }
}

/**
 * Attempts to remove redundant attributes from hyperblocks.
 *
 * This can be done if the attribute is not needed for classification,
 * for example if everything is seperable by x1, then we don't need to keep x2.
 *
 * Details are discussed further in "Fully Explainable Classification Models Using Hyperblocks", 2025. Ryan Gallagher, Austin Snyder, Boris Kovalerchuk
 * @param hyper_blocks
 * @param data
 * @param attributeOrderings
 */
void Simplifications::removeUselessAttr(std::vector<HyperBlock>& hyper_blocks, std::vector<std::vector<std::vector<float>>>& data, std::vector<std::vector<int>>& attributeOrderings) {
    int FIELD_LENGTH = data[0][0].size();

    // Prepare host data by flattening your data structures.
    auto fMinMaxResult =  DataUtil::flatMinMaxNoEncode(hyper_blocks, FIELD_LENGTH);
    auto fDataResult =  DataUtil::flattenDataset(data);

    // Build host arrays from the flattened results:
    std::vector<float> mins = fMinMaxResult[0];
    std::vector<float> maxes = fMinMaxResult[1];
    int minMaxLen = static_cast<int>(mins.size());

    std::vector<int> blockEdges(fMinMaxResult[2].size());
    for (size_t i = 0; i < fMinMaxResult[2].size(); i++) {
        blockEdges[i] = static_cast<int>(fMinMaxResult[2][i]);
    }
    int numBlocks = static_cast<int>(hyper_blocks.size());

    std::vector<int> blockClasses(fMinMaxResult[3].size());
    for (size_t i = 0; i < fMinMaxResult[3].size(); i++) {
        blockClasses[i] = static_cast<int>(fMinMaxResult[3][i]);
    }

    std::vector<int> intervalCounts(fMinMaxResult[4].size());
    for (size_t i = 0; i < fMinMaxResult[4].size(); i++) {
        intervalCounts[i] = static_cast<int>(fMinMaxResult[4][i]);
    }

    // Create flags array (initialize to 0).
    std::vector<char> attrRemoveFlags(hyper_blocks.size() * FIELD_LENGTH, 0);

    // Prepare the dataset.
    std::vector<float> dataset = fDataResult[0];
    int numPoints = static_cast<int>(dataset.size() / FIELD_LENGTH);

    std::vector<int> classBorder(fDataResult[1].size());
    for (size_t i = 0; i < fDataResult[1].size(); i++) {
        classBorder[i] = static_cast<int>(fDataResult[1][i]);
    }
    int numClasses = static_cast<int>(hyper_blocks.size());

    std::vector<int> attributeOrderingsFlattened(attributeOrderings.size() * FIELD_LENGTH, 0);
    for (int i = 0; i < attributeOrderings.size(); i++) {
        copy(attributeOrderings[i].begin(), attributeOrderings[i].end(),
            attributeOrderingsFlattened.begin() + i * FIELD_LENGTH);
    }

    // Device pointers.
    float* d_mins = nullptr;
    float* d_maxes = nullptr;
    int* d_intervalCounts = nullptr;
    int* d_blockEdges = nullptr;
    int* d_blockClasses = nullptr;
    char* d_attrRemoveFlags = nullptr;
    float* d_dataset = nullptr;
    int* d_classBorder = nullptr;
    int *d_attributeOrderingsFlattened = nullptr;

    // Allocate device memory.
    hipMalloc((void**)&d_mins, mins.size() * sizeof(float));
    hipMalloc((void**)&d_maxes, maxes.size() * sizeof(float));
    hipMalloc((void**)&d_intervalCounts, intervalCounts.size() * sizeof(int));
    hipMalloc((void**)&d_blockEdges, blockEdges.size() * sizeof(int));
    hipMalloc((void**)&d_blockClasses, blockClasses.size() * sizeof(int));
    hipMalloc((void**)&d_attrRemoveFlags, attrRemoveFlags.size() * sizeof(char));
    hipMalloc((void**)&d_dataset, dataset.size() * sizeof(float));
    hipMalloc((void**)&d_classBorder, classBorder.size() * sizeof(int));
    hipMalloc((void**)&d_attributeOrderingsFlattened, attributeOrderingsFlattened.size() * sizeof(int));

    // Copy host data to device.
    hipMemcpy(d_mins, mins.data(), mins.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_maxes, maxes.data(), maxes.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_intervalCounts, intervalCounts.data(), intervalCounts.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blockEdges, blockEdges.data(), blockEdges.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blockClasses, blockClasses.data(), blockClasses.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_attrRemoveFlags, attrRemoveFlags.data(), attrRemoveFlags.size() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_dataset, dataset.data(), dataset.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_classBorder, classBorder.data(), classBorder.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_attributeOrderingsFlattened, attributeOrderingsFlattened.data(), attributeOrderingsFlattened.size() * sizeof(int), hipMemcpyHostToDevice);

    // Determine execution configuration.
    int blockSize;
    int gridSize;

    hipError_t err = hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, mergerHyperBlocks, 0, 0);
    gridSize = (numBlocks + blockSize - 1) / blockSize;

    // Launch the kernel.
    removeUselessAttributes<<<gridSize, blockSize>>>(d_mins, d_maxes, d_intervalCounts, minMaxLen, d_blockEdges, numBlocks, d_blockClasses, d_attrRemoveFlags, FIELD_LENGTH, d_dataset, numPoints, d_classBorder, numClasses, d_attributeOrderingsFlattened);
    hipDeviceSynchronize();

    // Copy results from device (flags) back to host.
    hipMemcpy(attrRemoveFlags.data(), d_attrRemoveFlags, attrRemoveFlags.size() * sizeof(char), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_mins);
    hipFree(d_maxes);
    hipFree(d_intervalCounts);
    hipFree(d_blockEdges);
    hipFree(d_blockClasses);
    hipFree(d_attrRemoveFlags);
    hipFree(d_dataset);
    hipFree(d_classBorder);
    hipFree(d_attributeOrderingsFlattened);

    // Update the hyper_blocks based on the flags.
    for (size_t hb = 0; hb < hyper_blocks.size(); hb++) {
        HyperBlock &block = hyper_blocks[hb];
        // For each attribute in the block (assumes FIELD_LENGTH attributes per block)
        for (int attr = 0; attr < FIELD_LENGTH; attr++) {
            int index = hb * FIELD_LENGTH + attr;
            if (attrRemoveFlags[index] == 1) {
                // Remove the attribute intervals and reset to default values.
                if (attr < block.minimums.size() && attr < block.maximums.size()) {
                    block.minimums[attr].clear();
                    block.maximums[attr].clear();
                    block.minimums[attr].push_back(0.0f);
                    block.maximums[attr].push_back(1.0f);
                }
            }
        }
    }
}


vector<int> Simplifications::runSimplifications(vector<HyperBlock> &hyperBlocks, vector<vector<vector<float>>> &trainData, vector<vector<int>> &bestAttributeOrderings){
    int FIELD_LENGTH = trainData[0][0].size();
    int runCount = 0;
    int totalClauses = 0;
    int updatedClauses = 0;

    Simplifications::removeUselessAttr(hyperBlocks, trainData, bestAttributeOrderings);

    do{
        // set our count of what we have to start
        totalClauses = updatedClauses;
        runCount++; // counter so we can show how many iterations this took.

        // simplification functions
        Simplifications::removeUselessBlocks(trainData, hyperBlocks);

        // count how many we have after simplifications.
        updatedClauses = 0;
        for(HyperBlock &hyperBlock : hyperBlocks) {
            for(int i = 0; i < FIELD_LENGTH; i++){
                if (hyperBlock.minimums[i][0] == 0 && hyperBlock.maximums[i][0] == 1.0f){
                    continue;
                }
                else
                    updatedClauses += hyperBlock.minimums[i].size();
            }
        }

    // iteratively call the simplifications until we don't remove any more clauses.
    } while(updatedClauses != totalClauses);
    return { runCount, totalClauses };
}
